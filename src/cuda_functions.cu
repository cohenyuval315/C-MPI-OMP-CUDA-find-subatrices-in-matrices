#include "hip/hip_runtime.h"
#include <stdio.h>
#include "cuda_functions.h"
#include "config.h"
#include "struct.h"

void cuda_null_check(void* ptr , const char* msg){
    if(ptr == NULL){
        printf("%s\n",msg);
        exit(0);
    }
}

void CUDA_error_check(hipError_t error, const char* msg){
    if (error != hipSuccess)
	{
		printf("%s; Cuda Error:= %s (%d)\n",msg, hipGetErrorString(error), error);
    	exit(0);
  	}
}

__device__ double CUDA_diff(double matrix_element, double submatrix_element){
    return abs((matrix_element - submatrix_element) /matrix_element);
}

__host__ void CUDA_matrix_to_GPU(Matrix mat, double **devMatrixPtr) 
{
	int size = mat.rows * mat.cols;
	hipError_t error = hipSuccess;
	error = hipMalloc(devMatrixPtr, size * sizeof(double));
    CUDA_error_check(error,"fail to allocate memory for matrix ptr in GPU");
  	error = hipMemcpy(*devMatrixPtr, mat.data, size * sizeof(double), hipMemcpyHostToDevice);
    CUDA_error_check(error,"fail to copy memory for matrix ptr in GPU");
}

void CUDA_check_matrix_in_GPU(Matrix mat, double* devMatrixPtr){
    hipError_t error = hipSuccess;
    int size = mat.cols * mat.rows;
    double arr[size];
    error = hipMemcpy(arr,devMatrixPtr,size * sizeof(double),hipMemcpyDeviceToHost);
    CUDA_error_check(error,"fail to copy matrix from device to host");
}

double*  CUDA_get_matrix_from_device(double* arr , Matrix mat, double* devMatrixPtr){
    hipError_t error = hipSuccess;
    int size = mat.cols * mat.rows;
    arr = (double*)malloc(size* sizeof(double));
    cuda_null_check(arr,"fail to allocate memory for matrix");
    CUDA_check_matrix_in_GPU(mat,devMatrixPtr);
    error = hipMemcpy(arr,devMatrixPtr,size * sizeof(double),hipMemcpyDeviceToHost);
    CUDA_error_check(error,"fail to copy matrix from device to host");
    return arr;
}

void CUDA_check_array_in_GPU(double* devMatrixPtr, int size){
    hipError_t error = hipSuccess;
    double arr[size];
    error = hipMemcpy(arr,devMatrixPtr,size * sizeof(double),hipMemcpyDeviceToHost);
    CUDA_error_check(error,"fail to copy matrix from device to host");
}

double*  CUDA_get_array_from_device(double* devMatrixPtr, int size){
    hipError_t error = hipSuccess;
    double* arr = (double*)malloc(size* sizeof(double));
    cuda_null_check(arr,"fail to allocate memory for arr");
    CUDA_check_array_in_GPU(devMatrixPtr, size);
    error = hipMemcpy(arr,devMatrixPtr,size * sizeof(double),hipMemcpyDeviceToHost);
    CUDA_error_check(error,"fail to copy arr from device to host");
    return arr;
}

__host__ void CUDA_free_matrix_from_GPU(double **devMatrixPtr) 
{
	hipError_t error = hipSuccess;
	error = hipFree(*devMatrixPtr);
    CUDA_error_check(error,"fail to free matrix ptr memory from GPU");
}

__global__ void CUDA_calculate_matching_array_kernel(double* devMatchingValueArrayPtr, double* devMatrixPtr, double* devSubmatrixPtr, int matrix_rows, int matrix_cols, int submatrix_rows, int submatrix_cols) {
    int block_x = blockIdx.x;
    int block_y = blockIdx.y;
    int thread_x = threadIdx.x;
    int thread_y = threadIdx.y;
    int stride_x = blockDim.x * gridDim.x;
    int stride_y = blockDim.y * gridDim.y;
    int match_rows = matrix_rows - submatrix_rows + 1;
    int match_cols = matrix_cols - submatrix_cols + 1;

    for (int i = block_y * blockDim.y + thread_y; i < match_rows; i += stride_y) {
        for (int j = block_x * blockDim.x + thread_x; j < match_cols; j += stride_x) {
            int index = i * match_cols + j;
            double matching_value = 0.0;
            for (int k = 0; k < submatrix_rows; k++) {
                for (int l = 0; l < submatrix_cols; l++) {
                    int matrix_index = (i + k) * matrix_cols + j + l;
                    int submatrix_index = k * submatrix_cols + l;
                    double diff = CUDA_diff(devMatrixPtr[matrix_index], devSubmatrixPtr[submatrix_index]);
                    matching_value += diff;
                }
            }
            devMatchingValueArrayPtr[index] = matching_value;
        }
    }
}

double* CUDA_get_matrix_matching_array_in_matrix(double* devMatrixPtr,double *devSubmatrixPtr, Matrix mat,Matrix submatrix){
   
    if( submatrix.rows * submatrix.cols > mat.rows * mat.cols){
        return nullptr;
    }
    
    hipError_t error = hipSuccess;
    double *devMatchingValueArrayPtr;
    int num_threads_rows = mat.rows;
    int num_threads_cols = mat.cols;
    int grid_block_rows = num_threads_rows / CUDA_THREADS_DIM + ((num_threads_rows % CUDA_THREADS_DIM) != 0); 
    int grid_block_cols = num_threads_cols / CUDA_THREADS_DIM + ((num_threads_cols % CUDA_THREADS_DIM) != 0); 
    int match_value_arr_rows_size = mat.rows - submatrix.rows + 1;
    int match_value_arr_cols_size = mat.cols - submatrix.cols + 1;
    int total_num_positions = match_value_arr_rows_size * match_value_arr_cols_size;


    // host to device

    error = hipMalloc(&devMatchingValueArrayPtr, total_num_positions * sizeof(double));
    CUDA_error_check(error, "fail to allocate memory for matching value array pointer in GPU");

    dim3 dimGrid(grid_block_cols,grid_block_rows);
    dim3 dimBlock(CUDA_THREADS_DIM,CUDA_THREADS_DIM);

    CUDA_calculate_matching_array_kernel<<<dimGrid,dimBlock>>>(devMatchingValueArrayPtr,devMatrixPtr,devSubmatrixPtr,mat.rows,mat.cols,submatrix.rows,submatrix.cols);
    double* arr = CUDA_get_array_from_device(devMatchingValueArrayPtr,total_num_positions);
    error = hipFree(devMatchingValueArrayPtr);
    CUDA_error_check(error, "failed to free matching value array pointer memory from GPU");
    return arr;
}


